#include <stdio.h>
#include "NuclearEos.h"


void MemFree_NuclearEoS() {
   

   for (int t=0; t<NUC_TABLE_NPTR; t++)
   {
      if ( d_EoS_Table[t] != NULL ) 
      {
         hipFree( d_EoS_Table[t] );
         d_EoS_Table[t] = NULL;
      }
   }
   
   hipFree( d_energy_shift );
   d_energy_shift = NULL;
   hipFree( d_nrho );
   d_nrho = NULL;
   hipFree( d_neps );
   d_neps = NULL;
   hipFree( d_nye );
   d_nye = NULL;
   hipFree( d_nmode );
   d_nmode = NULL;

   hipFree( d_Rand_Vars );
   d_Rand_Vars = NULL;

}

