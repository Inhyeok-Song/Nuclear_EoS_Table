#include <stdio.h>
#include "NuclearEos.h"


void MemFree_NuclearEoS() {
   

   for ( int t=0; t<EOS_NTABLE_MAX; t++ )
   {
      if ( d_EoS_Table[t] != NULL ) 
      {
         hipFree( d_EoS_Table[t] );
         d_EoS_Table[t] = NULL;
      }
   }
   
   hipFree( d_energy_shift );
   d_energy_shift = NULL;
   hipFree( d_nrho );
   d_nrho = NULL;
   hipFree( d_neps );
   d_neps = NULL;
   hipFree( d_nye );
   d_nye = NULL;
   hipFree( d_nmode );
   d_nmode = NULL;

   for ( int t=0; t<NUC_TABLE_NPTR; t++ )
   { 
      if ( d_Rand_Vars != NULL )
      {
         hipFree( d_Rand_Vars[t] );
         d_Rand_Vars[t] = NULL;
      }
   }

}